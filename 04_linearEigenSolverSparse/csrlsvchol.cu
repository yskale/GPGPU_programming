#include "hip/hip_runtime.h"
// nvcc -arch=sm_70 -lcublas -lcusolver -lcusparse csrlsvchol.cu
// solve Ax=b, with Cholesky factorization for positive definite Hermitian (symmetry) matrix
// support both host & device execution, this is device version
// ref: https://stackoverflow.com/questions/30060067/cusolverspdcsrlsvlu-or-qr-method-using-cuda
#include "include/csr.hpp"
#include <limits>

#define __TIME_BEGIN hipEventRecord(start);
#define __TIME_END              \
    hipEventRecord(stop);      \
    hipEventSynchronize(stop); \
    hipEventElapsedTime(&elapsedTime, start, stop);

// #define SHOW_MATRIX
// #define DOUBLE_FP_CASE
#ifdef DOUBLE_FP_CASE
typedef double fp;
#else
typedef float fp;
#endif

const fp sparselevel = 0.5;
const int N = 1000;
constexpr int matSize = N * N;
fp *matA_h, *vecb_h, *resx_h;
fp *vecb_d, *resx_d;
int singularity = 0;
csrMat<fp> csrA_h;
csrMat<fp> csrA_d;
hipsolverSpHandle_t cusolverH = NULL;
hipsparseMatDescr_t descrA = NULL;
hipEvent_t start, stop;
float elapsedTime;

void print_matrix(const fp *arr, int M, int N)
{
    for (int i = 0; i < M; i++)
    {
        // std::cout << "row " << i << ": ";
        std::cout << std::fixed;
        for (int j = 0; j < N; j++)
        {
            std::cout << arr[j * M + i] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

void resources_init()
{
    matA_h = new fp[matSize]();
    vecb_h = new fp[N]();
    resx_h = new fp[N]();
    hipMalloc((void **)&vecb_d, N * sizeof(fp));
    hipMalloc((void **)&resx_d, N * sizeof(fp));

    // ======================================================================================
    // create a tmp random matrix
    fp *matTmp_h = new fp[matSize]();
    for (int i = 0; i < matSize; i++)
    {
        matTmp_h[i] = rand() / (fp)RAND_MAX * 1.0;
        if (rand() / (fp)RAND_MAX * 1.0 < sparselevel) // make the matrix become sparse
            matTmp_h[i] = 0.0;
    }

    // create positive definite Hermitian (symmetry) matrix, https://cplusplus.com/forum/general/257711/
    fp maxVal = std::numeric_limits<fp>::min();
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            matA_h[j * N + i] = 0;
            for (int k = 0; k < N; k++)
            {
                matA_h[j * N + i] += matTmp_h[k * N + i] * matTmp_h[k * N + j];
            }
            maxVal = std::max(maxVal, matA_h[j * N + i]);
        }
    }
    // normalization
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            matA_h[j * N + i] /= maxVal;
        }
    }
    delete[] matTmp_h;
    // ======================================================================================

    for (int i = 0; i < N; i++)
        vecb_h[i] = rand() / (fp)RAND_MAX * 1.0;

    int numNozeroA = 0;
    for (int i = 0; i < matSize; i++)
    {
        if (std::fabs(matA_h[i]) > 0.0)
            numNozeroA++;
    }

    csrA_h.init(numNozeroA, N, N, memType::host);
    csrA_h.dense2csrHost(matA_h);
#ifdef SHOW_MATRIX
    csrA_h.printCsrMatrixHost();
    csrA_h.printCsrFormHost();
#endif
    csrA_d.init(numNozeroA, N, N, memType::device);
    csrA_d.copyFromHost(csrA_h);

    memset(resx_h, 0, N * sizeof(fp));
    hipMemcpy(vecb_d, vecb_h, N * sizeof(fp), hipMemcpyHostToDevice);

    hipsolverSpCreate(&cusolverH);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipEventCreate(&start);
    hipEventCreate(&stop);

#ifdef SHOW_MATRIX
    std::cout << "A = \n";
    print_matrix(matA_h, N, N);
    std::cout << "b = \n";
    print_matrix(vecb_h, N, 1);
#endif
}

void result_reset()
{
    memset(resx_h, 0, N * sizeof(fp));
    hipMemcpy(vecb_d, vecb_h, N * sizeof(fp), hipMemcpyHostToDevice);
    csrA_d.copyFromHost(csrA_h);
}

void resources_free()
{
    delete[] matA_h;
    delete[] vecb_h;
    delete[] resx_h;

    hipFree(vecb_d);
    hipFree(resx_d);

    hipsolverSpDestroy(cusolverH);
    hipsparseDestroyMatDescr(descrA);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();
}

void check_result(fp *matA, fp *resX, fp *vecb, int n)
{
    fp errorNorm = 0.0;
    for (int i = 0; i < n; i++)
    {
        fp sum = 0.0;
        for (int j = 0; j < n; j++)
        {
            sum += matA[j * n + i] * resX[j];
        }
        errorNorm += std::pow<fp, int>((sum - vecb[i]), 2);
    }
    errorNorm = std::pow<fp, fp>(errorNorm, 0.5f);
    std::cout << "error 2-norm = " << errorNorm << std::endl;
}

int main()
{
    resources_init();
    for (int i = 0; i < 10; i++)
    {
        result_reset();
        __TIME_BEGIN
#ifdef DOUBLE_FP_CASE
        hipsolverSpDcsrlsvchol(cusolverH, csrA_d.nA, csrA_d.numA, descrA, csrA_d.csrValA, csrA_d.csrRowPtrA, csrA_d.csrColIndA, vecb_d, 0.0, 0, resx_d, &singularity);
#else
        hipsolverSpScsrlsvchol(cusolverH, csrA_d.nA, csrA_d.numA, descrA, csrA_d.csrValA, csrA_d.csrRowPtrA, csrA_d.csrColIndA, vecb_d, 0.0, 0, resx_d, &singularity);
#endif
        __TIME_END
        std::cout << "No. " << i << " run, GPU calculation time = " << elapsedTime << "ms\n";
    }

    hipMemcpy(resx_h, resx_d, sizeof(fp) * N, hipMemcpyDeviceToHost);
#ifdef SHOW_MATRIX
    std::cout << "x = \n";
    print_matrix(resx_h, N, 1);
#endif
    if (singularity == -1)
        check_result(matA_h, resx_h, vecb_h, N);
    else
        std::cout << "A is not symmetric postive definite, singularity = " << singularity << std::endl;

    resources_free();
    return 0;
}