#include "hip/hip_runtime.h"
// nvcc -arch=sm_70 -lcublas -lcusolver -lcusparse csrlsvqr.cu
// solve Ax=b, with QR factorization, support both host & device execution, this is device version
// ref: https://stackoverflow.com/questions/30060067/cusolverspdcsrlsvlu-or-qr-method-using-cuda
#include "include/csr.hpp"

#define __TIME_BEGIN hipEventRecord(start);
#define __TIME_END              \
    hipEventRecord(stop);      \
    hipEventSynchronize(stop); \
    hipEventElapsedTime(&elapsedTime, start, stop);

// #define SHOW_MATRIX
// #define DOUBLE_FP_CASE
#ifdef DOUBLE_FP_CASE
typedef double fp;
#else
typedef float fp;
#endif

const fp sparselevel = 0.3;
const int N = 1000;
constexpr int matSize = N * N;
fp *matA_h, *vecb_h, *resx_h;
fp *vecb_d, *resx_d;
int singularity = 0;
csrMat<fp> csrA_h;
csrMat<fp> csrA_d;
hipsolverSpHandle_t cusolverH = NULL;
hipsparseMatDescr_t descrA = NULL;
hipEvent_t start, stop;
float elapsedTime;

void print_matrix(const fp *arr, int M, int N)
{
    for (int i = 0; i < M; i++)
    {
        // std::cout << "row " << i << ": ";
        std::cout << std::fixed;
        for (int j = 0; j < N; j++)
        {
            std::cout << arr[j * M + i] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

void resources_init()
{
    matA_h = new fp[matSize]();
    vecb_h = new fp[N]();
    resx_h = new fp[N]();
    hipMalloc((void **)&vecb_d, N * sizeof(fp));
    hipMalloc((void **)&resx_d, N * sizeof(fp));

    for (int i = 0; i < matSize; i++)
    {
        matA_h[i] = rand() / (fp)RAND_MAX * 1.0;
        if (rand() / (fp)RAND_MAX * 1.0 < sparselevel) // make the matrix become sparse
            matA_h[i] = 0.0;
    }

    for (int i = 0; i < N; i++)
        vecb_h[i] = rand() / (fp)RAND_MAX * 1.0;

    int numNozeroA = 0;
    for (int i = 0; i < matSize; i++)
    {
        if (std::fabs(matA_h[i]) > 0.0)
            numNozeroA++;
    }

    csrA_h.init(numNozeroA, N, N, memType::host);
    csrA_h.dense2csrHost(matA_h);
#ifdef SHOW_MATRIX
    csrA_h.printCsrMatrixHost();
    csrA_h.printCsrFormHost();
#endif
    csrA_d.init(numNozeroA, N, N, memType::device);
    csrA_d.copyFromHost(csrA_h);

    memset(resx_h, 0, N * sizeof(fp));
    hipMemcpy(vecb_d, vecb_h, N * sizeof(fp), hipMemcpyHostToDevice);

    hipsolverSpCreate(&cusolverH);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipEventCreate(&start);
    hipEventCreate(&stop);

#ifdef SHOW_MATRIX
    std::cout << "A = \n";
    print_matrix(matA_h, N, N);
    std::cout << "b = \n";
    print_matrix(vecb_h, N, 1);
#endif
}

void result_reset()
{
    memset(resx_h, 0, N * sizeof(fp));
    hipMemcpy(vecb_d, vecb_h, N * sizeof(fp), hipMemcpyHostToDevice);
    csrA_d.copyFromHost(csrA_h);
}

void resources_free()
{
    delete[] matA_h;
    delete[] vecb_h;
    delete[] resx_h;

    hipFree(vecb_d);
    hipFree(resx_d);

    hipsolverSpDestroy(cusolverH);
    hipsparseDestroyMatDescr(descrA);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();
}

void check_result(fp *matA, fp *resX, fp *vecb, int n)
{
    fp errorNorm = 0.0;
    for (int i = 0; i < n; i++)
    {
        fp sum = 0.0;
        for (int j = 0; j < n; j++)
        {
            sum += matA[j * n + i] * resX[j];
        }
        errorNorm += std::pow<fp, int>((sum - vecb[i]), 2);
    }
    errorNorm = std::pow<fp, fp>(errorNorm, 0.5f);
    std::cout << "error 2-norm = " << errorNorm << std::endl;
}

int main()
{
    resources_init();
    for (int i = 0; i < 10; i++)
    {
        result_reset();
        __TIME_BEGIN
#ifdef DOUBLE_FP_CASE
        hipsolverSpDcsrlsvqr(cusolverH, csrA_d.nA, csrA_d.numA, descrA, csrA_d.csrValA, csrA_d.csrRowPtrA, csrA_d.csrColIndA, vecb_d, 0.0, 0, resx_d, &singularity);
#else
        hipsolverSpScsrlsvqr(cusolverH, csrA_d.nA, csrA_d.numA, descrA, csrA_d.csrValA, csrA_d.csrRowPtrA, csrA_d.csrColIndA, vecb_d, 0.0, 0, resx_d, &singularity);
#endif
        __TIME_END
        std::cout << "No. " << i << " run, GPU calculation time = " << elapsedTime << "ms\n";
    }

    hipMemcpy(resx_h, resx_d, sizeof(fp) * N, hipMemcpyDeviceToHost);
#ifdef SHOW_MATRIX
    std::cout << "x = \n";
    print_matrix(resx_h, N, 1);
#endif
    if(singularity == -1)
        check_result(matA_h, resx_h, vecb_h, N);
    else
        std::cout << "A is uninvertible, singularity = " << singularity << std::endl;

    resources_free();
    return 0;
}