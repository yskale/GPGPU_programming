#include "hip/hip_runtime.h"
// nvcc -arch=sm_70 -lcublas -lcusolver -lcusparse csrlsvqrHost.cu
// solve Ax=b, with QR factorization, support both host & device execution, this is host version
// ref: https://stackoverflow.com/questions/30060067/cusolverspdcsrlsvlu-or-qr-method-using-cuda
#include "include/csr.hpp"

#define __TIME_BEGIN hipEventRecord(start);
#define __TIME_END              \
    hipEventRecord(stop);      \
    hipEventSynchronize(stop); \
    hipEventElapsedTime(&elapsedTime, start, stop);

// #define SHOW_MATRIX
// #define DOUBLE_FP_CASE
#ifdef DOUBLE_FP_CASE
typedef double fp;
#else
typedef float fp;
#endif

const fp sparselevel = 0.3;
const int N = 1000;
constexpr int matSize = N * N;
fp *matA_h, *vecb_h, *resx_h;
int singularity = 0;
csrMat<fp> csrA_h;
hipsolverSpHandle_t cusolverH = NULL;
hipsparseMatDescr_t descrA = NULL;
hipEvent_t start, stop;
float elapsedTime;

void print_matrix(const fp *arr, int M, int N)
{
    for (int i = 0; i < M; i++)
    {
        // std::cout << "row " << i << ": ";
        std::cout << std::fixed;
        for (int j = 0; j < N; j++)
        {
            std::cout << arr[j * M + i] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

void resources_init()
{
    matA_h = new fp[matSize]();
    vecb_h = new fp[N]();
    resx_h = new fp[N]();

    for (int i = 0; i < matSize; i++)
    {
        matA_h[i] = rand() / (fp)RAND_MAX * 1.0;
        if (rand() / (fp)RAND_MAX * 1.0 < sparselevel) // make the matrix become sparse
            matA_h[i] = 0.0;
    }

    for (int i = 0; i < N; i++)
        vecb_h[i] = rand() / (fp)RAND_MAX * 1.0;

    int numNozeroA = 0;
    for (int i = 0; i < matSize; i++)
    {
        if (std::fabs(matA_h[i]) > 0.0)
            numNozeroA++;
    }

    csrA_h.init(numNozeroA, N, N, memType::host);
    csrA_h.dense2csrHost(matA_h);
#ifdef SHOW_MATRIX
    csrA_h.printCsrMatrixHost();
    csrA_h.printCsrFormHost();
#endif

    memset(resx_h, 0, N * sizeof(fp));

    hipsolverSpCreate(&cusolverH);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipEventCreate(&start);
    hipEventCreate(&stop);

#ifdef SHOW_MATRIX
    std::cout << "A = \n";
    print_matrix(matA_h, N, N);
    std::cout << "b = \n";
    print_matrix(vecb_h, N, 1);
#endif
}

void result_reset()
{
    memset(resx_h, 0, N * sizeof(fp));
}

void resources_free()
{
    delete[] matA_h;
    delete[] vecb_h;
    delete[] resx_h;

    hipsolverSpDestroy(cusolverH);
    hipsparseDestroyMatDescr(descrA);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();
}

void check_result(fp *matA, fp *resX, fp *vecb, int n)
{
    fp errorNorm = 0.0;
    for (int i = 0; i < n; i++)
    {
        fp sum = 0.0;
        for (int j = 0; j < n; j++)
        {
            sum += matA[j * n + i] * resX[j];
        }
        errorNorm += std::pow<fp, int>((sum - vecb[i]), 2);
    }
    errorNorm = std::pow<fp, fp>(errorNorm, 0.5f);
    std::cout << "error 2-norm = " << errorNorm << std::endl;
}

int main()
{
    resources_init();
    for (int i = 0; i < 10; i++)
    {
        result_reset();
        __TIME_BEGIN
#ifdef DOUBLE_FP_CASE
        cusolverSpDcsrlsvqrHost(cusolverH, csrA_h.nA, csrA_h.numA, descrA, csrA_h.csrValA, csrA_h.csrRowPtrA, csrA_h.csrColIndA, vecb_h, 0.0, 0, resx_h, &singularity);
#else
        cusolverSpScsrlsvqrHost(cusolverH, csrA_h.nA, csrA_h.numA, descrA, csrA_h.csrValA, csrA_h.csrRowPtrA, csrA_h.csrColIndA, vecb_h, 0.0, 0, resx_h, &singularity);
#endif
        __TIME_END
        std::cout << "No. " << i << " run, CPU calculation time = " << elapsedTime << "ms\n";
    }

#ifdef SHOW_MATRIX
    std::cout << "x = \n";
    print_matrix(resx_h, N, 1);
#endif
    if(singularity == -1)
        check_result(matA_h, resx_h, vecb_h, N);
    else
        std::cout << "A is uninvertible, singularity = " << singularity << std::endl;

    resources_free();
    return 0;
}